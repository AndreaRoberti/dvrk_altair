/* -*- Mode: C++; tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 4 -*-    */
/* ex: set filetype=cpp softtabstop=4 shiftwidth=4 tabstop=4 cindent expandtab: */

/*
  $Id$

  Author(s):  Min Yang Jung & Balazs Vagvolgyi
  Created on: 2010-05-24

  (C) Copyright 2010 Johns Hopkins University (JHU), All Rights
  Reserved.

--- begin cisst license - do not edit ---

This software is provided "as is" under an open source license, with
no warranty.  The complete license can be found in license.txt and
http://www.cisst.org/cisst/license.txt.

--- end cisst license ---

*/

#include <stdio.h>
#include <cutil_inline.h>


// CUDA configuration
const int ThreadCountPerBlock = 1 << 7; // 128 threads per block
//const int BlockCount = 1 << 5; // Warp size: 32

extern "C" bool cisst_cudaInitialize()
{
    int count = 0;

    hipGetDeviceCount(&count);
    if (count == 0) {
        printf("There is no device supporting CUDA.\n");
        return false;
    }

    int i = 0;
    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        printf("There is no device supporting CUDA.\n");
        return false;
    }
    hipSetDevice(i);

    return true;
}

extern "C" int cisst_cudaGetBlockSize(void)
{
	int device_count;
	int blocksize = 0;
	hipDeviceProp_t device_prop;
	
	hipGetDeviceCount(&device_count);

	// No Cuda Device Found
	if (device_count == 0)
		return 0;

    hipGetDeviceProperties(&device_prop, 0);

	// Limited By Shared Memory
	// 5 Blocks per MP
	// (2588+28) * 5 ==  13080 < (16384 shared memory of per MP)
	blocksize = ThreadCountPerBlock * 5 * device_prop.multiProcessorCount;

	return blocksize;
}


//////////////////////////////
// MEMORY HANDLER FUNCTIONS //
//////////////////////////////

extern "C" void* cisst_cudaMalloc(unsigned int size)
{
    void* ptr = 0;
    if (hipMalloc(&ptr, size) == hipSuccess) return ptr;
    return 0;
}

extern "C" bool cisst_cudaMemset(void* devPtr, unsigned char value, unsigned int size)
{
    if (devPtr == 0) return false;
    if (size == 0) return true;
    if (hipMemset(devPtr, (int)value, size) == hipSuccess) return true;
    return false;
}

extern "C" bool cisst_cudaMemcpyHostToDevice(void* devDestPtr, const void* hostSrcPtr, unsigned int size)
{
    if (devDestPtr == 0 || hostSrcPtr == 0) return false;
    if (size == 0) return true;
    if (hipMemcpy(devDestPtr, hostSrcPtr, size, hipMemcpyHostToDevice) != hipSuccess) return false;
    return true;
}

extern "C" bool cisst_cudaMemcpyDeviceToDevice(void* devDestPtr, const void* devSrcPtr, unsigned int size)
{
    if (devDestPtr == 0 || devSrcPtr == 0) return false;
    if (size == 0) return true;
    if (hipMemcpy(devDestPtr, devSrcPtr, size, hipMemcpyDeviceToDevice) != hipSuccess) return false;
    return true;
}

extern "C" bool cisst_cudaMemcpyDeviceToHost(void* hostDestPtr, const void* devSrcPtr, unsigned int size)
{
    if (hostDestPtr == 0 || devSrcPtr == 0) return false;
    if (size == 0) return true;
    if (hipMemcpy(hostDestPtr, devSrcPtr, size, hipMemcpyDeviceToHost) != hipSuccess) return false;
    return true;
}

extern "C" bool cisst_cudaFree(void* devPtr)
{
    if (devPtr == 0) return false;
    hipFree(devPtr);
    hipDeviceReset();
    return true;
}

